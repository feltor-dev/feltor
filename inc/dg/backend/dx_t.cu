#include "hip/hip_runtime.h"
#include <iostream>

//#include <cusp/ell_matrix.h>

#include "blas.h"
#include "dx.cuh"
#include "evaluation.cuh"
#include "typedefs.cuh"
#include "weights.cuh"

const double lx = 2*M_PI;
dg::direction dir = dg::centered;

/*
double function( double x) { return sin(x);}
double derivative( double x) { return cos(x);}
bc bcx = PER;
*/

double function (double  x) {return x*(x-2*M_PI)*exp(x);}
double derivative( double x) { return (2.*x-2*M_PI)*exp(x) + function(x);}
dg::bc bcx = dg::DIR;


/*
double function( double x) { return cos(x);}
double derivative( double x) { return -sin(x);}
bc bcx = NEU;
*/

/*
double function( double x) { return sin(3./4.*x);}
double derivative( double x) { return 3./4. * sin(3./4.*x);}
dg::bc bcx = dg::DIR_NEU;
*/

/*
double function( double x) { return cos(3./4.*x);}
double derivative( double x) { return -3./4.*sin(3./4.*x);}
bc bcx = NEU_DIR;
*/

int main ()
{
    unsigned int n = 3;
    unsigned int N = 40;

    std::cout << "Note the supraconvergence!\n";
    std::cout << "Type in n an Nx!\n";
    std::cin >> n>> N;
    std::cout << "# of Legendre nodes " << n <<"\n";
    std::cout << "# of cells          " << N <<"\n";
    dg::Grid1d<double> g( 0, lx, n, N);
    const double hx = lx/(double)N;
  cusp::ell_matrix< int, double, cusp::host_memory> hm = dg::create::dx_symm_normed<double>( n, N, hx, bcx);
//  cusp::ell_matrix< int, double, cusp::host_memory> hm = dg::create::dx_minus_normed<double>( n, N, hx, bcx);
//  cusp::ell_matrix< int, double, cusp::host_memory> hm = dg::create::dx_plus_normed<double>( n, N, hx, bcx);
    dg::HVec hv = dg::evaluate( function, g);
    dg::HVec hw = hv;
    const dg::HVec hu = dg::evaluate( derivative, g);


//    std::cout << "Input vector: " << std::endl;
//    for(dg::HVec::iterator it = hv.begin(); it != hv.end(); it++)
//    {
//        std::cout << *it << "  ";
//    }
//    std::cout << std::endl;

    dg::blas2::symv( hm, hv, hw);

//    std::cout << "True solution:" << std::endl;
//    for(unsigned int i = 0; i < hu.size(); i++)
//    {
//        std::cout << hu[i] << " ";
//    }
//    std::cout << std::endl;

//    std::cout << "Our solution:" << std::endl;
//    for(unsigned int i = 0; i < hu.size(); i++)
//    {
//        std::cout << hw[i] << " ";
//    }
//    std::cout << std::endl;


    dg::blas1::axpby( 1., hu, -1., hw);
    //
    std::cout << "Distance to true solution (cusp_matrix): "<<sqrt(dg::blas2::dot( dg::create::weights(g), hw) )<<"\n";

    dg::dx_matrix hm_2(n, N, hx, bcx, dir);
    hv = dg::evaluate(function, g);
    hw = hv;
    dg::blas2::symv(hm_2, hv, hw);
    dg::blas1::axpby(1., hu, -1., hw);

    std::cout << "Distance to true solution (dx_matrix): "<<sqrt(dg::blas2::dot( dg::create::weights(g), hw) )<<"\n";

    //for periodic bc | dirichlet bc
    //n = 1 -> p = 2      2
    //n = 2 -> p = 1      1
    //n = 3 -> p = 3      3
    //n = 4 -> p = 3      3
    //n = 5 -> p = 5      5


    
    return 0;
}
