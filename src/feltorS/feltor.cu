#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"
#include "file/read_input.h"

#include "feltor.cuh"
#include "parameters.h"
#include "probes.h"



/*
   - reads parameters from input.txt or any other given file, 
   - integrates the Feltor - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/


int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v2;
    std::stringstream title;
    if( argc == 1)
    {
        try{
            v = file::read_input("input.txt");
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else if( argc == 2)
    {
        try{
            v = file::read_input(argv[1]);
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const eule::Parameters p( v);
    p.display( std::cout);

    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow(  v2[2]*v2[3]*p.lx/p.ly, v2[1]*v2[4], "");
    draw::RenderHostData render( v2[1], v2[2]);



    //////////////////////////////////////////////////////////////////////////

    //Make grid
     dg::Grid2d<double > grid( 0., p.lx, 0.,p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);  
    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::DMatrix, dg::DVec, dg::DVec > feltor( grid, p); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //initial perturbation
    //dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Gaussian init0( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp);
//     dg::BathRZ init0(8, 8, 1, 0.0, 0.0, 30., 2., p.amp);
//     solovev::ZonalFlow init0(p, gp);
//     dg::CONSTANT init0( 0.);
//      dg::Vortex init0(  p.posX*p.lx, p.posY*p.ly, 0, p.sigma, p.amp);   
    //background profile
//     solovev::Nprofile prof(p, gp); //initial background profile
//     dg::CONSTANT prof(p.bgprofamp );
    //
//     dg::LinearX prof(-p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);
//     dg::SinProfX prof(p.nprofileamp, p.bgprofamp,M_PI/(2.*p.lx));
    dg::ExpProfX prof(p.nprofileamp, p.bgprofamp,p.ln);
//     const dg::DVec prof =  dg::LinearX( -p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);
//     dg::TanhProfX prof(p.lx*p.solb,p.lx/10.,-1.0,p.bgprofamp,p.nprofileamp); //<n>
    std::vector<dg::DVec> y0(2, dg::evaluate( prof, grid)), y1(y0); 
    

    //no field aligning
    y1[1] = dg::evaluate( init0, grid);
    dg::blas1::pointwiseDot(y1[1], y0[1],y1[1]); //<n>*ntilde
    
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni = <n> + <n>*ntilde
    dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); //initialize ni-1
//     dg::blas1::pointwiseDot(rolkar.damping(),y0[1], y0[1]); //damp with gaussprofdamp
    std::cout << "intiialize ne" << std::endl;
    feltor.initializene( y0[1], y0[0]);    
    std::cout << "Done!\n";


    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    std::cout << "intiialize karniadakis" << std::endl;
    karniadakis.init( feltor, rolkar, y0, p.dt);
    std::cout << "Done!\n";

    dg::DVec dvisual( grid.size(), 0.);
    dg::DVec dvisual2( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual),avisual(hvisual);
    dg::HMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    
    const double mass0 = feltor.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0., diff = 0.;
    
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    
    dg::DVec xprobecoords(7,1.);
    for (unsigned i=0;i<7; i++) {
        xprobecoords[i] = p.lx/8.*(1+i) ;
    }
    const dg::DVec yprobecoords(7,p.ly/2.);
    probes<dg::DMatrix, dg::DVec> pro(xprobecoords,yprobecoords,grid);
    while ( !glfwWindowShouldClose( w ))
    {

        hvisual = y0[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), (double)-1e14, thrust::maximum<double>() );
//         colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() = 1.0;
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ne / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ne-1 / " << colors.scalemin()<<"\t";

        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //draw ions
        //thrust::transform( y1[1].begin(), y1[1].end(), dvisual.begin(), dg::PLUS<double>(-0.));//ne-1
        hvisual = y0[1];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        //colors.scalemin() = 1.0;        
//         colors.scalemin() = -colors.scalemax();        
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ni / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ni-1 / " << colors.scalemin()<<"\t";

        render.renderQuad(visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        
        //draw potential
        //transform to Vor
//        dvisual=feltor.potential()[0];
//        dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
//        hvisual = y1[1];
         hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );

        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );

//         //colors.scalemin() = 1.0;        
//          colors.scalemin() = -colors.scalemax();        
//          colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"Potential / "<< colors.scalemax() << " " << colors.scalemin()<<"\t";

        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw potential
        //transform to Vor
        dvisual=feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
        hvisual = y1[1];
         //hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        //colors.scalemin() = 1.0;        
//          colors.scalemin() = -colors.scalemax();        
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"Omega / "<< colors.scalemax()<< " "<< colors.scalemin()<<"\t";

        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);


     
           
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
            step++;
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass_blob0<<"\t";
            E1 = feltor.energy();
            diff = (E1 - E0)/p.dt; //
            double diss = feltor.energy_diffusion( );
            double coupling = feltor.coupling();
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << 
//                          " Ne_p  = " << feltor.probe_vector()[0][0] << 
//                          " Phi_p = " << feltor.probe_vector()[1][0] << 
                         " Ga_nex= " << feltor.radial_transport() <<
                         " Coupling= " << coupling <<
                         " Accuracy: "<< 2.*fabs((diff-diss)/(diff+diss))<<
                         " d E/dt = " << diff <<
                         " Lambda =" << diss <<  std::endl;
            E0 = E1;
        }
        dg::blas1::transform( y0[0], dvisual, dg::PLUS<>(+(p.bgprofamp + p.nprofileamp))); //npe = N+1
        dvisual2 = feltor.potential()[0];
        pro.fluxes(time,  dvisual,dvisual2);
        pro.profiles(time,dvisual,dvisual2);
//         p.profiles
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
