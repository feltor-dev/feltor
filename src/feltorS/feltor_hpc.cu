#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG



#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"

#include "dg/backend/interpolation.cuh"
#include "file/read_input.h"
#include "file/nc_utilities.h"

#include "feltor.cuh"
#include "parameters.h"


/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

const unsigned k = 3;//!< a change in k needs a recompilation

int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v3;
    std::string input, geom;
    if( argc != 3)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        try{
            input = file::read_file( argv[1]);
            v = file::read_input( argv[1]);
        }catch( toefl::Message& m){
            m.display();
            std::cout << input << std::endl;
            return -1;
        }
    }
    const eule::Parameters p( v);
    p.display( std::cout);

    //Make grid
    dg::Grid2d<double> grid( 0., p.lx, 0., p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    dg::Grid2d<double> grid_out( 0., p.lx, 0., p.ly, p.n_out, p.Nx_out, p.Ny_out, p.bc_x, p.bc_y);  
    // Grid for radial probe location. This is used only in netcdf output, probe positioning is still hard-coded
    dg::Grid1d<double> grid_probe(0, p.lx, 1, 8, p.bc_x);
    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::DMatrix, dg::DVec, dg::DVec > feltor( grid, p); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::DMatrix, dg::DVec, dg::DVec > rolkar( grid, p);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //initial perturbation
    //dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Gaussian init0(p.posX * p.lx, p.posY * p.ly, p.sigma, p.sigma, p.amp);
//     dg::BathRZ init0(16, 16, 1, 2.0, 2.0, 30.0, 5.0, p.amp);
//     solovev::ZonalFlow init0(p, gp);
//     dg::CONSTANT init0( 0.);
    
    //background profile
//     solovev::Nprofile prof(p, gp); //initial background profile
//     dg::CONSTANT prof(p.bgprofamp );
    //
//     dg::LinearX prof(-p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);
//     dg::SinProfX prof(p.nprofileamp, p.bgprofamp,M_PI/(2.*p.lx));
        dg::ExpProfX prof(p.nprofileamp, p.bgprofamp,p.ln);
//     dg::TanhProfX prof(p.lx*p.solb,p.ln,-1.0,p.bgprofamp,p.nprofileamp); //<n>
//     dg::TanhProfX prof(p.lx*p.solb,p.lx/10.,-1.0,p.bgprofamp,p.nprofileamp); //<n>

//     const dg::DVec prof =  dg::LinearX( -p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);

    std::vector<dg::DVec> y0(2, dg::evaluate(prof, grid)), y1(y0); 
    //no field aligning
    y1[1] = dg::evaluate( init0, grid);
    dg::blas1::pointwiseDot(y1[1], y0[1], y1[1]);

    dg::blas1::axpby(1., y1[1], 1., y0[1]); //initialize ni
    dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); //initialize ni-1
//     dg::blas1::pointwiseDot(rolkar.damping(),y0[1], y0[1]); //damp with gaussprofdamp
    std::cout << "intiialize ne" << std::endl;
    feltor.initializene(y0[1], y0[0]);    
    std::cout << "Done!\n";


    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( feltor, rolkar, y0, p.dt);
//     feltor.energies( y0);//now energies and potential are at time 0
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[2], NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    int dim_ids_field[3], tvarID_field;
    err = file::define_dimensions(ncid, dim_ids_field, &tvarID_field, grid_out);
    err = nc_enddef(ncid);
    err = nc_redef(ncid);

    //field IDs
    std::vector<std::string> varname_fields;
    varname_fields.push_back("electrons"); varname_fields.push_back("ions"); varname_fields.push_back("potential"); varname_fields.push_back("vor");
    int dataIDs[4]; 
    for(unsigned i = 0; i < varname_fields.size(); i++)
        err = nc_def_var(ncid, varname_fields[i].data(), NC_DOUBLE, 3, dim_ids_field, &dataIDs[i]);
    //energy IDs, used for small time-step diagnostic
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    int energyID, massID, energyIDs[3], dissID, dEdtID, accuracyID, couplingID; 

    err = nc_def_var(ncid, "energy", NC_DOUBLE, 1, &EtimeID, &energyID);
    err = nc_def_var(ncid, "mass", NC_DOUBLE, 1, &EtimeID, &massID);
    err = nc_def_var(ncid, "dissipation", NC_DOUBLE, 1, &EtimeID, &dissID);
    err = nc_def_var(ncid, "dEdt", NC_DOUBLE, 1, &EtimeID, &dEdtID);
    err = nc_def_var(ncid, "accuracy", NC_DOUBLE, 1, &EtimeID, &accuracyID);
    err = nc_def_var(ncid, "Coupling", NC_DOUBLE, 1, &EtimeID, &couplingID);  

    std::string energies[3] = {"Se", "Si", "Uperp"}; 
    for(unsigned i = 0; i < 3; i++)
    {
        err = nc_def_var( ncid, energies[i].data(), NC_DOUBLE, 1, &EtimeID, &energyIDs[i]);
    }

   // Probe IDs
    std::vector<std::string> varname_probes;
    varname_probes.push_back("probe_ne"); varname_probes.push_back("probe_phi"); varname_probes.push_back("probe_Gamma_x");
    // Create x-dimension for probe 

    int ID_probes[3];
    int dim_ids_probe[2];
    dim_ids_probe[0] = EtimeID;
    //dim_ids_probe[1] = 
    file :: define_dimension(ncid, "X_probe", &dim_ids_probe[1], dg::evaluate(dg::LinearX(1.0, 0), grid_probe).data(), 8);
    for(unsigned i = 0; i < varname_probes.size(); i++)
    {
        err = nc_def_var(ncid, varname_probes[i].data(), NC_DOUBLE, 2, dim_ids_probe, &ID_probes[i]);
    } 
    
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    std::cout << "First output ... \n";
    size_t start[3] = {0, 0, 0};
    size_t count[3] = {1, grid_out.n() * grid_out.Ny(), grid_out.n() * grid_out.Nx()};
    dg::DVec transfer( dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::DMatrix interpolate = dg::create::interpolation( grid_out, grid); 
    for( unsigned i=0; i<2; i++)
    {
        dg::blas2::gemv( interpolate, y0[i], transferD);
        transferH = transferD;//transfer to host
        err = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    //pot
    transfer = feltor.potential()[0];
    dg::blas2::symv( interpolate, transfer, transferD);
    transferH = transferD;//transfer to host

    err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
    //Vor
    transfer = feltor.potential()[0];
    dg::blas2::gemv( rolkar.laplacianM(), transfer, y1[1]);            
    dg::blas2::symv( interpolate,y1[1], transferD);
    transferH = transferD;//transfer to host
    err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data() );
    double time = 0;

    // Probes
    size_t count_probes[2] = {1, grid_probe.n() * grid_probe.N()};
    size_t start_probes[2] = {0, 0};
    feltor.update_probes();
    dg::HVec probe_value(feltor.get_probe_vector()[0]);
    err = nc_put_vara_double( ncid, ID_probes[0], start_probes, count_probes, probe_value.data());
    probe_value = feltor.get_probe_vector()[1];
    err = nc_put_vara_double( ncid, ID_probes[1], start_probes, count_probes, probe_value.data());
    probe_value = feltor.get_probe_vector()[2];
    err = nc_put_vara_double(ncid, ID_probes[2], start_probes, count_probes, probe_value.data());

    err = nc_put_vara_double( ncid, tvarID_field, start, count, &time);
    err = nc_put_vara_double( ncid, EtimevarID, start, count, &time);

    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    double energy0 = feltor.energy(), mass0 = feltor.mass(), E0 = energy0, mass = mass0, E1 = 0.0, dEdt = 0., diss = 0., accuracy=0.;
//     double Nep=feltor.probe_vector()[0][0];
//     double phip=feltor.probe_vector()[1][0];
    double Nep=0.;
    double phip=0.;
    double radtrans = feltor.radial_transport();
    double coupling = feltor.coupling();
    std::vector<double> evec = feltor.energy_vector();
    err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &energy0);
    err = nc_put_vara_double( ncid, massID, Estart, Ecount, &mass0);
    for( unsigned i=0; i<3; i++)
        err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);

    err = nc_put_vara_double( ncid, dissID, Estart, Ecount, &diss);
    err = nc_put_vara_double( ncid, dEdtID, Estart, Ecount, &dEdt);
    //probe
    err = nc_put_vara_double(ncid, ID_probe_Ne, Estart, Ecount_probe, probe_Ne.data());
    err = nc_put_vara_double(ncid, ID_probe_phi, Estart, Ecount_probe, probe_phi.data());
    err = nc_put_vara_double(ncid, ID_probe_Gamma, Estart, Ecount, probe_Gamma.data());
    err = nc_put_vara_double( ncid, couplingID, Estart, Ecount, &coupling);
    err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount, &accuracy);

    err = nc_close(ncid);
    std::cout << "First write successful!\n";

    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Timer t;
    t.tic();
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=1; i<=p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                err = nc_close(ncid);
                return -1;
            }
            step++;
            time+=p.dt;
            feltor.update_probes();
            Estart[0] = step;
            E1 = feltor.energy(), mass = feltor.mass(), diss = feltor.energy_diffusion();
            dEdt = (E1 - E0)/p.dt; 
            E0 = E1;
            accuracy = 2.*fabs( (dEdt-diss)/(dEdt + diss));
            evec = feltor.energy_vector();
//             Nep =feltor.probe_vector()[0][0];
//             phip=feltor.probe_vector()[1][0];
            radtrans = feltor.radial_transport();
            coupling= feltor.coupling();
            err = nc_open(argv[2], NC_WRITE, &ncid);
            err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
            err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &E1);
            err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass);
            for( unsigned i=0; i<3; i++)
            {
                err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);
            }
            err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
            err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
            err = nc_put_vara_double( ncid, couplingID, Estart, Ecount,&coupling);    
            err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);

            start_probes[0] = step;

            probe_value = feltor.get_probe_vector()[0];
            err = nc_put_vara_double( ncid, ID_probes[0], start_probes, count_probes, probe_value.data());
            probe_value = feltor.get_probe_vector()[1];
            err = nc_put_vara_double( ncid, ID_probes[1], start_probes, count_probes, probe_value.data());
            probe_value = feltor.get_probe_vector()[2];
            err = nc_put_vara_double( ncid, ID_probes[2], start_probes, count_probes, probe_value.data());

            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass0<<"\t";
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout <<" d E/dt = " << dEdt <<" Lambda = " << diss << " -> Accuracy: "<< accuracy << "\n";
            err = nc_close(ncid);
        }
#ifdef DG_BENCHMARK
        ti.toc();
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
        //////////////////////////write fields////////////////////////
        start[0] = i;
        err = nc_open(argv[2], NC_WRITE, &ncid);
        for(unsigned j = 0; j < 2; j++)
        {
            dg::blas2::symv( interpolate, y0[j], transferD);
            transferH = transferD;//transfer to host
            err = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data());
        }
        transfer = feltor.potential()[0];
        dg::blas2::symv( interpolate, transfer, transferD);
        transferH = transferD;//transfer to host
        err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
        transfer = feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), transfer, y1[1]);            
        dg::blas2::symv( interpolate,y1[1], transferD);
        transferH = transferD;//transfer to host
        err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data());

        err = nc_put_vara_double(ncid, tvarID_field, start, count, &time);
        err = nc_close(ncid);
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;
}

